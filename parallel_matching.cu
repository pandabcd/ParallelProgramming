#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<fstream>
#include<vector>

using namespace std;

#define num_threads 10
// #define num_edges 700000
// #define num_vertices1 10000
// #define num_vertices2 10000

// #define num_edges 1000000
// #define num_vertices1 1000
// #define num_vertices2 1000


#define lli long long int

// #define num_edges 2998468
// #define num_vertices1 100000
// #define num_vertices2 100000



const lli num_edges = 4;
const lli num_vertices1 = 2;
const lli num_vertices2 = 2;

__device__ int d_flat_adj_list[2*num_edges];
__device__ int d_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
__device__ int d_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
__device__ int d_list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep

__device__ bool d_is_matched_edge[(num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)] = {0} ;     // Adjacency matrix (1-indexed)
__device__ bool d_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
__device__ int d_partner_vertex[num_vertices1 + num_vertices2 + 1];
__device__ int d_visited[num_vertices1 + num_vertices2 + 1] = {0};
__device__ int d_bfs_parent[num_vertices1 +  num_vertices2 + 1];
__device__ bool d_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

__device__ int d_frontier[num_vertices1 + num_vertices2+1] = {0};
__device__ int d_next_frontier[num_vertices1+num_vertices2+1] = {0};



int *h_flat_adj_list;
int *h_degree;
int * h_list_ptr;
int *h_list_ptr_copy;

bool *h_is_matched_edge;
bool *h_is_matched_vertex;
int *h_partner_vertex;
int *h_visited;
int *h_bfs_parent;
bool *h_is_parent_change;

int fc = num_vertices1;
// int num_aug_paths = 0;

int *h_frontier;
int *h_next_frontier;


__device__ 
int get_is_matched_edge(int i, int j){
	return d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ];
}

__device__ 
void set_is_matched_edge(int i, int j, int value){
	d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ] = value;
}
__device__
void clear_visited(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_visited[vertex1] = 0;
	}
}


__device__ 
void clear_bfs_parent(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_bfs_parent[vertex1] = vertex1;
	}
}

__device__ 
void initialise_partner_vertex(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_partner_vertex[vertex1] = -1;
	}
}

__device__ 
void clear_is_parent_change(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_is_parent_change[vertex1] = -1;
	}
}

__device__ 
void copy_frontier(int *my_frontier, int *my_next_frontier){
	
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
		my_frontier[i] = my_next_frontier[i];
	}
}

__device__ 
void clear_frontier(int *my_frontier, int *my_next_frontier ){
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
			my_frontier[i] = 0;
			my_next_frontier[i] = 0;
	}
}
__device__
void vertex_disjoint_bfs(int binary_level, int vertex){
	// int frontier_element = vertex;
	// printf("Frontier element: %d \n", frontier_element );
	if(!d_frontier[vertex]){
		return;
	}

	// my_frontier[vertex] = 1;
	d_frontier[vertex] = 1;


	// Iterate all frontier elements
	while(frontier_element!=-1){
	
		int vertex = frontier_element;
		
		// Make this atomic
		d_visited[vertex] = true;
		
		// cout << "Frontier: " << frontier_element << endl;
		// cout << "Continuining for vertex: " << vertex << endl;
		
		bool found_path = false;
		int start_edge = d_list_ptr[vertex];
		int end_edge = d_list_ptr[vertex + 1]; 
		
		// cout << "Start-End edge " <<  start_edge << " " << end_edge  << endl;
		printf ("Start-End edge %d %d \n", start_edge, end_edge);
		for(int j=start_edge;j<end_edge;j++){
			if(found_path)
				break;


			int neighbor = d_flat_adj_list[j];

			

			int visited = atomicExch(&d_visited[neighbor], 1);

			if(!visited){
				// We want to alternate between unmatched and matched edges, otherwise we ignore
				d_visited[neighbor] = true;
				// cout << "Processing: " << vertex << " " << neighbor << endl;
				// exit(0);
				d_bfs_parent[neighbor] = vertex;

				if( binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==1 ){
					// next_frontier.push_back(neighbor);
					d_next_frontier[neighbor] = 1;
				}

				// is_matched_vertex is implicitly true since the edge is matched
				// In level 1, we are only interested in matched edges
				else if( binary_level==1 && get_is_matched_edge(vertex, neighbor)==1 ){
					// next_frontier.push_back(neighbor);
					d_next_frontier[neighbor] = 1;
					// If I have found a path to the next level; I have to break
					// found_path = 1;
					return;
				}

				// Changing parent change only for this node
				else if(binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==0){
					// cout << "Found a aug. path with " << neighbor << " with parent: " << vertex << endl;
					printf("Found a aug. path with %d with parent: %d \n", neighbor, vertex);
					d_is_parent_change[neighbor] = 1;
					// num_aug_paths++ ;
					// remove this return so that multiple paths can be found 
					return;
				}
			}
		}

		// frontier_element = get_frontier_element(vertex);
		// Getting next frontier element
		for(int x = vertex+1; x <=num_vertices1 + num_vertices2; x++){
			if(d_frontier[x]==1){
				frontier_element = x;
				break;
			}
		}
		break;
	}
	copy_frontier(d_frontier, d_next_frontier);
	// Only working for one level for now
	// bfs(binary_level = !binary_level);

}


__global__
void vertex_disjoint_bfs_util(){
	
	clear_visited();
	clear_bfs_parent();
	clear_is_parent_change();

	// int *my_frontier = new int[num_vertices1+num_vertices2+1];
	// int *my_next_frontier = new int[num_vertices1+num_vertices2+1];
	// clear_frontier(my_frontier, my_next_frontier );

	initialise_partner_vertex();
	//Can add fairness here

	
	// int num_aug_paths = 1000;

	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex = tid+1;
	if(vertex > num_vertices1)
		return;

	if(!d_visited[vertex] && !d_is_matched_vertex[vertex]){
		d_frontier[vertex] = 1;
		vertex_disjoint_bfs(0, vertex);
		__syncthreads();
	}

	printf("Working \n" );

	// if(num_aug_paths > 0){
	// 	update_matchings();
	// }

	// return num_aug_paths;
}



int main(){
	h_is_matched_edge = (bool *)calloc( (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1), sizeof(bool));

	h_flat_adj_list = (int *)malloc(2*num_edges*sizeof(int));
	h_degree = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_list_ptr = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_list_ptr_copy = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_is_matched_vertex = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_partner_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_visited = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_bfs_parent = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_is_parent_change = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_next_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));

	// Add a check for null memory

	memset(h_degree, 0, num_vertices1 + num_vertices2 +1);
	// memset(h_is_matched_edge, 0, (num_vertices1 + num_vertices2 +1)*(num_vertices1+num_vertices2+1));
	memset(h_is_matched_vertex, 0, num_vertices1 + num_vertices2 +1);
	memset(h_visited, 0, num_vertices1 + num_vertices2 +1);
	memset(h_is_parent_change, 0, num_vertices1 + num_vertices2 +1);
	memset(h_frontier, 0, num_vertices1 + num_vertices2 +1);
	memset(h_next_frontier, 0, num_vertices1 + num_vertices2 +1);



	// to and from of edges
	// int h_edges_u[num_edges], h_edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	int *h_edges_u, *h_edges_v;
	h_edges_u = (int *)malloc((num_edges)*sizeof(int));
	h_edges_v = (int *)malloc((num_edges)*sizeof(int));


	ifstream fin;
    fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    // fin.open("random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt", ios::in);
    int u, v;

    // cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
    		// cout << i << endl;
            fin >> u >> v;
            h_edges_u[i] = u;
            h_edges_v[i] = v;
            h_degree[u]++;
            h_degree[v]++;
    }

    cout << "Done reading edges" << endl;

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    h_list_ptr[1] = 0;
    h_list_ptr_copy[1] = h_list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	h_list_ptr[i] = h_list_ptr[i-1] + h_degree[i-1];
    	h_list_ptr_copy[i] = h_list_ptr[i];
    }
    h_list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    h_list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position

    
    for(int i=0;i<num_edges;i++){
    	h_flat_adj_list[h_list_ptr_copy[h_edges_u[i]]] = h_edges_v[i];
    	h_flat_adj_list[h_list_ptr_copy[h_edges_v[i]]] = h_edges_u[i];
    	h_list_ptr_copy[h_edges_u[i]]++;
    	h_list_ptr_copy[h_edges_v[i]]++;
    }
    

	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_edge), h_is_matched_edge, (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), h_flat_adj_list, 2*num_edges*sizeof(int)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_degree), h_degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), h_list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_vertex), h_is_matched_vertex, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_visited), h_visited, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_frontier), h_frontier, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	



    // for(int i=1;i<=num_vertices1+num_vertices2;i++){
    // 	for(int j=1;j<=num_vertices1+num_vertices2+1;j++){
    // 		h_is_matched_edge[j*num_vertices2 + i] = 0;
    // 	}
    // }
    // sleep(20000);

    // initialise_partner_vertex();
    // cout << "Partner vertex initialized " << endl;
  	

  	// for(int i=1;i<=num_vertices1+num_vertices2;i++){
  	// 	cout << h_degree[i] << " ";
  	// }

  	// for(int i=0;i<2*num_edges;i++){
  	// 	cout << h_flat_adj_list[i] << " ";
  	// }
  	// cout << endl;
   //  for(int i=0;i<=num_vertices1+num_vertices2;i++){
  	// 	cout << h_list_ptr[i] << " ";
  	// // }

    // cout << " ------------------------" <<endl;
    // for(int i=1;i<=num_vertices1;i++){
    // 	for(int j=1;j<=num_vertices2;j++){
    // 		get_is_matched_edge(i,j);
    // 	}
    // }


    // cout << get_frontier_element(9265);
  	// int x = check_matching();

  	cout << "Matching checked " << endl;

  	vertex_disjoint_bfs_util<<<1, num_threads>>>();
  	hipDeviceSynchronize();
  


}