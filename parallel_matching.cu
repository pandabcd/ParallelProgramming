
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<fstream>
#include<time.h>
#include<sys/time.h>
#include<string>

using namespace std;

#define num_threads 1000
// #define num_edges 700000
// #define num_vertices1 10000
// #define num_vertices2 10000

// #define num_edges 1000000
// #define num_vertices1 1000
// #define num_vertices2 1000


#define lli long long int

// const lli num_edges = 2998468;
// const lli num_vertices1 = 100000;
// const lli num_vertices2 = 100000;

const lli num_edges = 1000000;
const lli num_vertices1 = 1000;
const lli num_vertices2 = 1000;

// const lli num_edges = 700000;
// const lli num_vertices1 = 10000;
// const lli num_vertices2 = 10000;



// const lli num_edges = 291;
// const lli num_vertices1 = 100;
// const lli num_vertices2 = 100;

// const lli num_edges = 291;
// const lli num_vertices1 = 100;
// const lli num_vertices2 = 100;

__device__ int d_flat_adj_list[2*num_edges];
__device__ int d_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
__device__ int d_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
__device__ int d_list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep

// __device__ bool d_is_matched_edge[(num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)] = {0} ;     // Adjacency matrix (1-indexed)
__device__ bool d_matched_edge[2*num_edges];					// Tells for every edge in the list if the edge is matched or not
__device__ bool d_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
__device__ int d_partner_vertex[num_vertices1 + num_vertices2 + 1];
__device__ int d_visited[num_vertices1 + num_vertices2 + 1] = {0};
__device__ int d_bfs_parent[num_vertices1 +  num_vertices2 + 1];
__device__ bool d_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

__device__ int d_frontier[num_vertices1 + num_vertices2+1] = {0};
__device__ int d_next_frontier[num_vertices1+num_vertices2+1] = {0};

__device__ int num_aug_paths = 10000000;						//Any number not equal to 0 works



int *h_flat_adj_list;
int *h_degree;
int * h_list_ptr;
int *h_list_ptr_copy;

bool *h_matched_edge;
bool *h_is_matched_vertex;
int *h_partner_vertex;
int *h_visited;
int *h_bfs_parent;
bool *h_is_parent_change;

int fc = num_vertices1;
// int num_aug_paths = 0;

int *h_frontier;
int *h_next_frontier;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
bool get_matched_edge(int x, int y){
	int vertex = x;
	int start_edge = d_list_ptr[vertex];
	int end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i]==y){
			return d_matched_edge[i];
		}
	}
	printf("Error! Querying for an edge which is not present \n");
	return -1;
}

__device__
void set_matched_edge(int x, int y, int value){
	bool edge_present = false;
	int vertex = x;
	int start_edge = d_list_ptr[vertex];
	int end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i] == y){
			d_matched_edge[i] = value;
			edge_present = true;
			break;
		}
	}
	
	vertex = y;
	start_edge = d_list_ptr[vertex];
	end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i] == x){
			d_matched_edge[i] = value;
			edge_present = true;
			break;
		}
	}

	if(!edge_present){
		printf("Error! Querying for an edge which is not present \n");
	}
}

// __device__ 
// int get_is_matched_edge(int i, int j){
// 	return d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ];
// }

void print_matchings(){
	cout << "Matchings: " << endl;
    for(int i=1;i<=num_vertices1+num_vertices2; i++){
    	cout<< i << " " << h_partner_vertex[i] << endl;
    }
}

int get_matched_edge_h(int x, int y){
	int vertex = x;
	int start_edge = h_list_ptr[vertex];
	int end_edge = h_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(h_flat_adj_list[i] == y){
			return h_matched_edge[i];
		}
	}
	cout << "Error! Querying for an edge which is not present";
	exit(0);
}

// __device__ 
// void set_is_matched_edge(int i, int j, int value){
// 	d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ] = value;
// }

__device__
void match_edges(int u, int v){
	set_matched_edge(u,v,1);
	set_matched_edge(v,u,1);
	d_is_matched_vertex[u] = 1;
	d_is_matched_vertex[v] = 1;
	d_partner_vertex[u] = v;
	d_partner_vertex[v] = u;

}

// Unmatching edges also unmatches the vertices since the graph is a matching
__device__
void unmatch_edges(int u, int v){
	set_matched_edge(u,v,0);
	set_matched_edge(v,u,0);
	if(d_partner_vertex[u]==v){
		d_is_matched_vertex[u] = 0;
		d_partner_vertex[u] = -1;
	}
	if(d_partner_vertex[v]==u){
		d_is_matched_vertex[v] = 0;
		d_partner_vertex[v] = -1;
	}
}

// Make this parallel
__device__
void update_matchings(){
	for(int i=1; i<=num_vertices1+num_vertices2; i++){
		int vertex = i;
		if(d_is_parent_change[vertex] == true){
			
			// cout << "Found aug. path till " << vertex << endl;
			// There should always be odd number of vertices in aug. path
			int path_length = 1;
			int parent = d_bfs_parent[vertex];
			while(parent!=vertex){
				// cout << vertex << " " <<parent << endl;
				if(path_length%2==1){
					match_edges(vertex, parent);
					// printf("Matching %d and %d \n", vertex, parent);
				}
				else{
					unmatch_edges(vertex, parent);
					// printf("Unmatching %d and %d \n", vertex, parent);
				}
				vertex =  d_bfs_parent[vertex];
				parent = d_bfs_parent[vertex];
				path_length++;
			}
		}
	}
}



__device__
void clear_visited(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_visited[vertex1] = 0;
	}
}


__device__ 
void clear_bfs_parent(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_bfs_parent[vertex1] = vertex1;
	}
}

__device__ 
void initialise_partner_vertex(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_partner_vertex[vertex1] = -1;
	}
}

__device__ 
void clear_is_parent_change(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_is_parent_change[vertex1] = -1;
	}
}

__device__ 
void copy_frontier(int *my_frontier, int *my_next_frontier){
	
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
		my_frontier[i] = my_next_frontier[i];
	}
}

__device__ 
void clear_frontier(int *my_frontier, int *my_next_frontier ){
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
			my_frontier[i] = 0;
			my_next_frontier[i] = 0;
	}
}
__device__
void vertex_disjoint_bfs(int binary_level, int vertex, int tid){
	// int frontier_element = vertex;
	// printf("Frontier element: %d \n", frontier_element );
	// if(!d_frontier[vertex]){
	// 	return;
	// }

	// my_frontier[vertex] = 1;
	// d_frontier[vertex] = 1;


	// Iterate all frontier elements
	// if(frontier_element!=-1){
	
		// int vertex = frontier_element;
		
		// if(vertex >= num_vertices1 + num_vertices2){
		// 		printf("Error");
		// 	}

		// Make this atomic
		int visited_self = atomicExch(&d_visited[vertex], 1);
		if(visited_self){
			return;
		}
		d_visited[vertex] = true;
		
		// cout << "Frontier: " << frontier_element << endl;
		// cout << "Continuining for vertex: " << vertex << endl;
		
		bool found_path = false;
		int start_edge = d_list_ptr[vertex];
		int end_edge = d_list_ptr[vertex + 1]; 
		
		for(int j=start_edge;j<end_edge;j++){
			if(found_path)
				break;


			int neighbor = d_flat_adj_list[j];


			if(neighbor > num_vertices1 + num_vertices2){
				printf("[%d]Error(neighbor out of range: vertex, neighbor : %d, %d \n", tid, vertex, neighbor);
			}
			// printf("[%d]Processing %d %d \n", tid, vertex, neighbor);
			// printf("[%d]Start-End %d %d %d \n",tid, j, start_edge, end_edge);

			int visited = atomicExch(&d_visited[neighbor], 1);

			if(!visited){
				// We want to alternate between unmatched and matched edges, otherwise we ignore
				d_visited[neighbor] = true;
				d_bfs_parent[neighbor] = vertex;

				if( binary_level==0 && get_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==1 ){
						d_next_frontier[neighbor] = 1;
				}

				// is_matched_vertex is implicitly true since the edge is matched
				// In level 1, we are only interested in matched edges
				else if( binary_level==1 && get_matched_edge(vertex, neighbor)==1 ){
					d_next_frontier[neighbor] = 1;
					// If I have found a path to the next level; I have to break
					// found_path = 1;
					return;
				}

				// Changing parent change only for this node
				else if(binary_level==0 && get_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==0){
					d_is_parent_change[neighbor] = 1;
					// We are not interested in the exact number but if it is greater than 0
					// atomicAdd(&num_aug_paths, 1);
					num_aug_paths++;
					return;
				}
			}
		// }

		// frontier_element = get_frontier_element(vertex);
		// Getting next frontier element
		// for(int x = vertex+1; x <=num_vertices1 + num_vertices2; x++){
		// 	if(d_frontier[x]==1){
		// 		frontier_element = x;
		// 		break;
		// 	}
		// }
	}
	copy_frontier(d_frontier, d_next_frontier);
	// Only working for one level for now
	vertex_disjoint_bfs(!binary_level, vertex, tid);

}


__global__
void vertex_disjoint_bfs_util(){

	// parallelise these functions
	clear_visited();
	clear_bfs_parent();
	clear_is_parent_change();

	// clear_frontier(my_frontier, my_next_frontier );
	initialise_partner_vertex();


	// int num_aug_paths = 1000;

	int tid = blockIdx.x*1024 + threadIdx.x;

	

	int vertex = tid+1;
	if(vertex > num_vertices1)
		return;

	if(vertex >=  num_vertices1+num_vertices2+1)
		printf("[%d] Error \n");

	// printf("[%d] Call \n", tid);
	// printf("Bid: %d ; Threadid: %d ; tid: %d \n", blockIdx.x, threadIdx.x, tid);
	// bool find_aug_paths = (num_aug_paths!=0);
	// while(find_aug_paths){
		// num_aug_paths = 0;
		if(!d_visited[vertex] && !d_is_matched_vertex[vertex]){
			d_frontier[vertex] = 1;
			vertex_disjoint_bfs(0, vertex, tid);
		}

		if(num_aug_paths > 0){
			update_matchings();
		}
		// else{
		// 	break;
		// }
	// }

	
		// num_aug_paths = -10;
	// return num_aug_paths;
}

int check_matching(){
	int total_matched = 0;
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		int vertex = i;
		int num_matched = 0;


		for(int j=h_list_ptr[i];j<h_list_ptr[i+1];j++){
			int neighbor = h_flat_adj_list[j];
			// cout << vertex << " " << neighbor << endl;
			if(get_matched_edge_h(vertex, neighbor)){
				num_matched++;
			}
		}


		if(num_matched==1){
			// cout << "Hi" << endl;
			total_matched++;
		}
		if(num_matched>1){
			cout << vertex << endl;
			cout << "Error! Not a matching!";
			exit(0);
		}
	}
	return total_matched/2;
}


int main(){

	struct timespec start, end;

	// h_is_matched_edge = (bool *)calloc( (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1), sizeof(bool));

	h_matched_edge = (bool *)calloc(2*num_edges, sizeof(bool));
	
	h_flat_adj_list = (int *)malloc(2*num_edges*sizeof(int));
	h_degree = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_list_ptr = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_list_ptr_copy = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_is_matched_vertex = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_partner_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_visited = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_bfs_parent = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_is_parent_change = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_next_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));

	// Add a check for null memory

	memset(h_degree, 0, num_vertices1 + num_vertices2 +1);
	// memset(h_is_matched_edge, 0, (num_vertices1 + num_vertices2 +1)*(num_vertices1+num_vertices2+1));
	memset(h_is_matched_vertex, 0, num_vertices1 + num_vertices2 +1);
	memset(h_visited, 0, num_vertices1 + num_vertices2 +1);
	memset(h_is_parent_change, 0, num_vertices1 + num_vertices2 +1);
	memset(h_frontier, 0, num_vertices1 + num_vertices2 +1);
	memset(h_next_frontier, 0, num_vertices1 + num_vertices2 +1);



	// to and from of edges
	// int h_edges_u[num_edges], h_edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	int *h_edges_u, *h_edges_v;
	h_edges_u = (int *)malloc((num_edges)*sizeof(int));
	h_edges_v = (int *)malloc((num_edges)*sizeof(int));


	ifstream fin;
	// string fileName = "FC_1000_1000.txt"; 
    // fin.open("random_100000_100000.txt", ios::in);
    fin.open("FC_1000_1000.txt", ios::in);
    // fin.open("random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt", ios::in);
    // cout << "random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt" <<endl;
    int u, v;

    // cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
    		// cout << i << endl;
            fin >> u >> v;
            h_edges_u[i] = u;
            h_edges_v[i] = v;
            h_degree[u]++;
            h_degree[v]++;
    }

    cout << "Done reading edges" << endl;

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    h_list_ptr[1] = 0;
    h_list_ptr_copy[1] = h_list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	h_list_ptr[i] = h_list_ptr[i-1] + h_degree[i-1];
    	h_list_ptr_copy[i] = h_list_ptr[i];
    }
    h_list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    h_list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position

    
    for(int i=0;i<num_edges;i++){
    	h_flat_adj_list[h_list_ptr_copy[h_edges_u[i]]] = h_edges_v[i];
    	h_flat_adj_list[h_list_ptr_copy[h_edges_v[i]]] = h_edges_u[i];
    	h_list_ptr_copy[h_edges_u[i]]++;
    	h_list_ptr_copy[h_edges_v[i]]++;
    }
    

    clock_gettime( CLOCK_REALTIME,&start);

	hipMemcpyToSymbol(HIP_SYMBOL(d_matched_edge), h_matched_edge, (2*num_edges)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), h_flat_adj_list, 2*num_edges*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_degree), h_degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), h_list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_vertex), h_is_matched_vertex, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_visited), h_visited, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_frontier), h_frontier, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);

	int h_num_aug_paths = 1000;
	
  	hipDeviceSynchronize();


  	while(h_num_aug_paths>0){
  		h_num_aug_paths = 0;
  		hipMemcpyToSymbol(HIP_SYMBOL(num_aug_paths), &h_num_aug_paths, (1)*sizeof(int),0,hipMemcpyHostToDevice);
	  	
	  	vertex_disjoint_bfs_util<<<10, 1024>>>();

	 //  	gpuErrchk( cudaPeekAtLastError() );
		// gpuErrchk( cudaDeviceSynchronize() );


	  	hipDeviceSynchronize();
	  	hipMemcpyFromSymbol(h_matched_edge, HIP_SYMBOL(d_matched_edge), sizeof(d_matched_edge),0,hipMemcpyDeviceToHost);
	  	hipMemcpyFromSymbol(h_partner_vertex, HIP_SYMBOL(d_partner_vertex), sizeof(d_partner_vertex),0,hipMemcpyDeviceToHost);
	  	
	  	hipMemcpyFromSymbol(&h_num_aug_paths, HIP_SYMBOL(num_aug_paths), sizeof(num_aug_paths),0,hipMemcpyDeviceToHost);

	  	printf("Number of augmenting paths(actual number may be higher): %d \n", h_num_aug_paths);
	  	break;
	}
  	clock_gettime( CLOCK_REALTIME,&end);
  	
  

  	int num_matches = check_matching();
  	// exit(0);
  	// print_matchings();

  	printf("Number of matchings: %d \n", num_matches);

  	double elapsed = (end.tv_sec-start.tv_sec)*1000000000 + end.tv_nsec-start.tv_nsec;
  	printf("Time elapsed %lf\n", elapsed/1e6);
	

  	hipDeviceSynchronize();
  


}