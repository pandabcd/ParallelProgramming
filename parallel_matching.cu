
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<fstream>
#include<time.h>
#include<sys/time.h>
#include<string>

using namespace std;

// #define num_threads 1000

#define lli long long int

int size[5] = {100, 500, 1000, 5000, 10000};

// int edges_2[5] = {200, 447, 1969, 4991, 200001};
int edges_8[5] = {801, 20000, 79580, 1999218, 8000000};


const lli num_edges = 8000000;
const lli num_vertices1 = 10000;
const lli num_vertices2 = 10000;


__device__ int d_flat_adj_list[2*num_edges];
__device__ int d_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
__device__ int d_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
__device__ int d_list_ptr_copy[num_vertices1+num_vertices2+2];    // 


__device__ bool d_matched_edge[2*num_edges];					// Tells for every edge in the list if the edge is matched or not
__device__ bool d_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
__device__ int d_partner_vertex[num_vertices1 + num_vertices2 + 1];
__device__ int d_visited[num_vertices1 + num_vertices2 + 1] = {0};
__device__ int d_bfs_parent[num_vertices1 +  num_vertices2 + 1];
__device__ bool d_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

__device__ int d_frontier[num_vertices1 + num_vertices2+1] = {0};
__device__ int d_next_frontier[num_vertices1+num_vertices2+1] = {0};

__device__ int num_aug_paths = 10000000;						//Any number not equal to 0 works



int *h_flat_adj_list;
int *h_degree;
int * h_list_ptr;
int *h_list_ptr_copy;

bool *h_matched_edge;
bool *h_is_matched_vertex;
int *h_partner_vertex;
int *h_visited;
int *h_bfs_parent;
bool *h_is_parent_change;

int fc = num_vertices1;
// int num_aug_paths = 0;

int *h_frontier;
int *h_next_frontier;


__device__
bool get_matched_edge(int x, int y){
	int vertex = x;
	int start_edge = d_list_ptr[vertex];
	int end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i]==y){
			return d_matched_edge[i];
		}
	}
	printf("Error! Querying for an edge which is not present \n");
	return -1;
}

__device__
void set_matched_edge(int x, int y, int value){
	bool edge_present = false;
	int vertex = x;
	int start_edge = d_list_ptr[vertex];
	int end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i] == y){
			d_matched_edge[i] = value;
			edge_present = true;
			break;
		}
	}
	
	vertex = y;
	start_edge = d_list_ptr[vertex];
	end_edge = d_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(d_flat_adj_list[i] == x){
			d_matched_edge[i] = value;
			edge_present = true;
			break;
		}
	}

	if(!edge_present){
		printf("Error! Querying for an edge which is not present \n");
	}
}

void print_matchings(){
	cout << "Matchings: " << endl;
    for(int i=1;i<=num_vertices1+num_vertices2; i++){
    	cout<< i << " " << h_partner_vertex[i] << endl;
    }
}

int get_matched_edge_h(int x, int y){
	int vertex = x;
	int start_edge = h_list_ptr[vertex];
	int end_edge = h_list_ptr[vertex + 1]; 
	for(int i = start_edge; i<end_edge;i++){
		if(h_flat_adj_list[i] == y){
			return h_matched_edge[i];
		}
	}
	cout << "Error! Querying for an edge which is not present";
	exit(0);
}


__device__
void match_edges(int u, int v){
	set_matched_edge(u,v,1);
	set_matched_edge(v,u,1);
	d_is_matched_vertex[u] = 1;
	d_is_matched_vertex[v] = 1;
	d_partner_vertex[u] = v;
	d_partner_vertex[v] = u;

}

// Unmatching edges also unmatches the vertices since the graph is a matching
__device__
void unmatch_edges(int u, int v){
	set_matched_edge(u,v,0);
	set_matched_edge(v,u,0);
	if(d_partner_vertex[u]==v){
		d_is_matched_vertex[u] = 0;
		d_partner_vertex[u] = -1;
	}
	if(d_partner_vertex[v]==u){
		d_is_matched_vertex[v] = 0;
		d_partner_vertex[v] = -1;
	}
}

// Make this parallel
__global__
void update_matchings(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	for(int i=tid; i<=num_vertices1+num_vertices2; i+=num_vertices1){
		int vertex = i;
		if(d_is_parent_change[vertex] == true){
			
			// cout << "Found aug. path till " << vertex << endl;
			// There should always be odd number of vertices in aug. path
			int path_length = 1;
			int parent = d_bfs_parent[vertex];
			while(parent!=vertex){
				// cout << vertex << " " <<parent << endl;
				if(path_length%2==1){
					match_edges(vertex, parent);
					// printf("Matching %d and %d \n", vertex, parent);
				}
				else{
					unmatch_edges(vertex, parent);
					// printf("Unmatching %d and %d \n", vertex, parent);
				}
				vertex =  d_bfs_parent[vertex];
				parent = d_bfs_parent[vertex];
				path_length++;
			}
		}
	}
}

__device__
void clear_visited(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_visited[vertex1] = 0;
	}
}

__device__ 
void clear_bfs_parent(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_bfs_parent[vertex1] = vertex1;
	}
}

__device__ 
void initialise_partner_vertex(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_partner_vertex[vertex1] = -1;
	}
}

__device__ 
void clear_is_parent_change(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_is_parent_change[vertex1] = -1;
	}
}

__device__ 
void copy_frontier(int *my_frontier, int *my_next_frontier){
	
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
		my_frontier[i] = my_next_frontier[i];
	}
}

__device__ 
void clear_frontier(int *my_frontier, int *my_next_frontier ){
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
			my_frontier[i] = 0;
			my_next_frontier[i] = 0;
	}
}

__device__
void vertex_disjoint_bfs(int binary_level, int vertex, int tid){
	
		int visited_self = atomicExch(&d_visited[vertex], 1);
		if(visited_self && binary_level==0){
			return;
		}
		d_visited[vertex] = true;
		
		
		bool found_path = false;
		int start_edge = d_list_ptr[vertex];
		int end_edge = d_list_ptr[vertex + 1]; 
		
		for(int j=start_edge;j<end_edge;j++){
			if(found_path)
				break;


			int neighbor = d_flat_adj_list[j];


			if(neighbor > num_vertices1 + num_vertices2){
				printf("[%d]Error(neighbor out of range: vertex, neighbor : %d, %d \n", tid, vertex, neighbor);
			}
			
			int visited = atomicExch(&d_visited[neighbor], 1);

			if(!visited){
				// We want to alternate between unmatched and matched edges, otherwise we ignore
				d_visited[neighbor] = true;
				d_bfs_parent[neighbor] = vertex;

				if( binary_level==0 && get_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==1 ){
						d_next_frontier[neighbor] = 1;
						if(binary_level==1)
							printf("Going odd %d \n", vertex);
						vertex_disjoint_bfs(!binary_level, neighbor, tid);
				}

				// In level 1, we are only interested in matched edges
				else if( binary_level==1 && get_matched_edge(vertex, neighbor)==1 ){
					d_next_frontier[neighbor] = 1;
					vertex_disjoint_bfs(!binary_level, neighbor, tid);
					return;
				}

				// Changing parent change only for this node
				else if(binary_level==0 && get_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==0){
					d_is_parent_change[neighbor] = 1;
					// atomicAdd(&num_aug_paths, 1);
					num_aug_paths++;
					return;
				}
			}
	}
}


__global__
void vertex_disjoint_bfs_util(){

	// parallelise these functions
	clear_visited();
	clear_bfs_parent();
	clear_is_parent_change();

	// clear_frontier(my_frontier, my_next_frontier );
	initialise_partner_vertex();
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex = tid+1;
	if(vertex > num_vertices1)
		return;

	if(vertex >=  num_vertices1+num_vertices2+1)
		printf("[%d] Error \n");

	if(!d_visited[vertex] && !d_is_matched_vertex[vertex]){
		d_frontier[vertex] = 1;
		vertex_disjoint_bfs(0, vertex, tid);
	}

}

int check_matching(){
	int total_matched = 0;
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		int vertex = i;
		int num_matched = 0;


		for(int j=h_list_ptr[i];j<h_list_ptr[i+1];j++){
			int neighbor = h_flat_adj_list[j];
			// cout << vertex << " " << neighbor << endl;
			if(get_matched_edge_h(vertex, neighbor)){
				num_matched++;
			}
		}


		if(num_matched==1){
			// cout << "Hi" << endl;
			total_matched++;
		}
		if(num_matched>1){
			cout << vertex << endl;
			cout << "Error! Not a matching!";
			exit(0);
		}
	}
	return total_matched/2;
}


int main(){

	struct timespec start, end;

	// h_is_matched_edge = (bool *)calloc( (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1), sizeof(bool));

	h_matched_edge = (bool *)calloc(2*num_edges, sizeof(bool));
	
	h_flat_adj_list = (int *)malloc(2*num_edges*sizeof(int));
	h_degree = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_list_ptr = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_list_ptr_copy = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_is_matched_vertex = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_partner_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_visited = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_bfs_parent = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_is_parent_change = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_next_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));

	// Add a check for null memory

	memset(h_degree, 0, num_vertices1 + num_vertices2 +1);
	// memset(h_is_matched_edge, 0, (num_vertices1 + num_vertices2 +1)*(num_vertices1+num_vertices2+1));
	memset(h_is_matched_vertex, 0, num_vertices1 + num_vertices2 +1);
	memset(h_visited, 0, num_vertices1 + num_vertices2 +1);
	memset(h_is_parent_change, 0, num_vertices1 + num_vertices2 +1);
	memset(h_frontier, 0, num_vertices1 + num_vertices2 +1);
	memset(h_next_frontier, 0, num_vertices1 + num_vertices2 +1);



	// to and from of edges
	// int h_edges_u[num_edges], h_edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	int *h_edges_u, *h_edges_v;
	h_edges_u = (int *)malloc((num_edges)*sizeof(int));
	h_edges_v = (int *)malloc((num_edges)*sizeof(int));


	ifstream fin;
	fin.open("random_10000_10000_high.txt", ios::in);
    int u, v;

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
    		// cout << i << endl;
            fin >> u >> v;
            h_edges_u[i] = u;
            h_edges_v[i] = v;
            h_degree[u]++;
            h_degree[v]++;
    }

    cout << "Done reading edges" << endl;

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    h_list_ptr[1] = 0;
    h_list_ptr_copy[1] = h_list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	h_list_ptr[i] = h_list_ptr[i-1] + h_degree[i-1];
    	h_list_ptr_copy[i] = h_list_ptr[i];
    }
    h_list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    h_list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position

    
    for(int i=0;i<num_edges;i++){
    	h_flat_adj_list[h_list_ptr_copy[h_edges_u[i]]] = h_edges_v[i];
    	h_flat_adj_list[h_list_ptr_copy[h_edges_v[i]]] = h_edges_u[i];
    	h_list_ptr_copy[h_edges_u[i]]++;
    	h_list_ptr_copy[h_edges_v[i]]++;
    }
    

    clock_gettime( CLOCK_REALTIME,&start);

	hipMemcpyToSymbol(HIP_SYMBOL(d_matched_edge), h_matched_edge, (2*num_edges)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), h_flat_adj_list, 2*num_edges*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_degree), h_degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), h_list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_vertex), h_is_matched_vertex, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_visited), h_visited, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_frontier), h_frontier, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);

	int h_num_aug_paths = 1000;
	
  	hipDeviceSynchronize();

  	while(h_num_aug_paths>0){
  		h_num_aug_paths = 0;
  		hipMemcpyToSymbol(HIP_SYMBOL(num_aug_paths), &h_num_aug_paths, (1)*sizeof(int),0,hipMemcpyHostToDevice);
	  	
	  	vertex_disjoint_bfs_util<<<10, 1024>>>();
	  	update_matchings<<<10, 1024>>>();
	  	hipDeviceSynchronize();	  	
	  	hipMemcpyFromSymbol(&h_num_aug_paths, HIP_SYMBOL(num_aug_paths), sizeof(num_aug_paths),0,hipMemcpyDeviceToHost);
	  	
	  	break;
	}
  	clock_gettime( CLOCK_REALTIME,&end);
  	hipMemcpyFromSymbol(h_matched_edge, HIP_SYMBOL(d_matched_edge), sizeof(d_matched_edge),0,hipMemcpyDeviceToHost);
  	hipMemcpyFromSymbol(h_partner_vertex, HIP_SYMBOL(d_partner_vertex), sizeof(d_partner_vertex),0,hipMemcpyDeviceToHost);
  	
  printf("Number of augmenting paths(actual number may be higher): %d \n", h_num_aug_paths);

  	int num_matches = check_matching();
  	
  	printf("Number of matchings: %d \n", num_matches);

  	double elapsed = (end.tv_sec-start.tv_sec)*1000000000 + end.tv_nsec-start.tv_nsec;
  	printf("Time elapsed %lf\n", elapsed/1e6);
	

  	hipDeviceSynchronize();
  
}