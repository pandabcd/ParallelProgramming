
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<fstream>
#include<vector>

using namespace std;

#define num_threads 100
// #define num_edges 700000
// #define num_vertices1 10000
// #define num_vertices2 10000

// #define num_edges 1000000
// #define num_vertices1 1000
// #define num_vertices2 1000


#define lli long long int

// #define num_edges 2998468
// #define num_vertices1 100000
// #define num_vertices2 100000



// const lli num_edges = 1000000;
// const lli num_vertices1 = 1000;
// const lli num_vertices2 = 1000;

const lli num_edges = 291;
const lli num_vertices1 = 100;
const lli num_vertices2 = 100;

__device__ int d_flat_adj_list[2*num_edges];
__device__ int d_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
__device__ int d_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
__device__ int d_list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep

__device__ bool d_is_matched_edge[(num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)] = {0} ;     // Adjacency matrix (1-indexed)
__device__ bool d_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
__device__ int d_partner_vertex[num_vertices1 + num_vertices2 + 1];
__device__ int d_visited[num_vertices1 + num_vertices2 + 1] = {0};
__device__ int d_bfs_parent[num_vertices1 +  num_vertices2 + 1];
__device__ bool d_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

__device__ int d_frontier[num_vertices1 + num_vertices2+1] = {0};
__device__ int d_next_frontier[num_vertices1+num_vertices2+1] = {0};

__device__ int num_aug_paths = 0;



int *h_flat_adj_list;
int *h_degree;
int * h_list_ptr;
int *h_list_ptr_copy;

bool *h_is_matched_edge;
bool *h_is_matched_vertex;
int *h_partner_vertex;
int *h_visited;
int *h_bfs_parent;
bool *h_is_parent_change;

int fc = num_vertices1;
// int num_aug_paths = 0;

int *h_frontier;
int *h_next_frontier;


__device__ 
int get_is_matched_edge(int i, int j){
	return d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ];
}

int get_is_matched_edge_h(int i, int j){
	return h_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ];
}

__device__ 
void set_is_matched_edge(int i, int j, int value){
	d_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ] = value;
}

__device__
void match_edges(int u, int v){
	set_is_matched_edge(u,v,1);
	set_is_matched_edge(v,u,1);
	d_is_matched_vertex[u] = 1;
	d_is_matched_vertex[v] = 1;
	d_partner_vertex[u] = v;
	d_partner_vertex[v] = u;

}

// Unmatching edges also unmatches the vertices since the graph is a matching
__device__
void unmatch_edges(int u, int v){
	set_is_matched_edge(u,v,0);
	set_is_matched_edge(v,u,0);
	if(d_partner_vertex[u]==v){
		d_is_matched_vertex[u] = 0;
		d_partner_vertex[u] = -1;
	}
	if(d_partner_vertex[v]==u){
		d_is_matched_vertex[v] = 0;
		d_partner_vertex[v] = -1;
	}
}

// Make this parallel
__device__
void update_matchings(){
	for(int i=1; i<=num_vertices1+num_vertices2; i++){
		int vertex = i;
		if(d_is_parent_change[vertex] == true){
			
			// cout << "Found aug. path till " << vertex << endl;
			// There should always be odd number of vertices in aug. path
			int path_length = 1;
			int parent = d_bfs_parent[vertex];
			while(parent!=vertex){
				// cout << vertex << " " <<parent << endl;
				if(path_length%2==1){
					match_edges(vertex, parent);
					// printf("Matching %d and %d \n", vertex, parent);
				}
				else{
					unmatch_edges(vertex, parent);
					// printf("Unmatching %d and %d \n", vertex, parent);
				}
				vertex =  d_bfs_parent[vertex];
				parent = d_bfs_parent[vertex];
				path_length++;
			}
		}
	}
}



__device__
void clear_visited(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_visited[vertex1] = 0;
	}
}


__device__ 
void clear_bfs_parent(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_bfs_parent[vertex1] = vertex1;
	}
}

__device__ 
void initialise_partner_vertex(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_partner_vertex[vertex1] = -1;
	}
}

__device__ 
void clear_is_parent_change(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1 + num_vertices2){
		d_is_parent_change[vertex1] = -1;
	}
}

__device__ 
void copy_frontier(int *my_frontier, int *my_next_frontier){
	
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
		my_frontier[i] = my_next_frontier[i];
	}
}

__device__ 
void clear_frontier(int *my_frontier, int *my_next_frontier ){
	for (int i=1;i<=num_vertices1+num_vertices2;i++){
			my_frontier[i] = 0;
			my_next_frontier[i] = 0;
	}
}
__device__
void vertex_disjoint_bfs(int binary_level, int vertex, int tid){
	int frontier_element = vertex;
	// printf("Frontier element: %d \n", frontier_element );
	if(!d_frontier[vertex]){
		return;
	}

	// my_frontier[vertex] = 1;
	d_frontier[vertex] = 1;


	// Iterate all frontier elements
	if(frontier_element!=-1){
	
		int vertex = frontier_element;
		
		// Make this atomic
		d_visited[vertex] = true;
		
		// cout << "Frontier: " << frontier_element << endl;
		// cout << "Continuining for vertex: " << vertex << endl;
		
		bool found_path = false;
		int start_edge = d_list_ptr[vertex];
		int end_edge = d_list_ptr[vertex + 1]; 
		
		// cout << "Start-End edge " <<  start_edge << " " << end_edge  << endl;
		// printf ("Start-End edge %d %d \n", start_edge, end_edge);
		for(int j=start_edge;j<end_edge;j++){
			if(found_path)
				break;


			int neighbor = d_flat_adj_list[j];

			// printf("[%d]Processing %d %d \n", tid, vertex, neighbor);

			int visited = atomicExch(&d_visited[neighbor], 1);

			if(!visited){
				// We want to alternate between unmatched and matched edges, otherwise we ignore
				d_visited[neighbor] = true;
				// cout << "Processing: " << vertex << " " << neighbor << endl;
				// printf("[%d]Processing %d %d \n", tid, vertex, neighbor);
				// exit(0);
				d_bfs_parent[neighbor] = vertex;

				if( binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==1 ){
					// next_frontier.push_back(neighbor);
					d_next_frontier[neighbor] = 1;
				}

				// is_matched_vertex is implicitly true since the edge is matched
				// In level 1, we are only interested in matched edges
				else if( binary_level==1 && get_is_matched_edge(vertex, neighbor)==1 ){
					// next_frontier.push_back(neighbor);
					d_next_frontier[neighbor] = 1;
					// If I have found a path to the next level; I have to break
					// found_path = 1;
					return;
				}

				// Changing parent change only for this node
				else if(binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && d_is_matched_vertex[neighbor]==0){
					// printf("Found a aug. path with %d with parent: %d \n", neighbor, vertex);
					d_is_parent_change[neighbor] = 1;
					num_aug_paths++ ;
					// remove this return so that multiple paths can be found 
					return;
				}
			}
		}

		// frontier_element = get_frontier_element(vertex);
		// Getting next frontier element
		for(int x = vertex+1; x <=num_vertices1 + num_vertices2; x++){
			if(d_frontier[x]==1){
				frontier_element = x;
				break;
			}
		}
	}
	copy_frontier(d_frontier, d_next_frontier);
	// Only working for one level for now
	// bfs(binary_level = !binary_level);

}


__global__
void vertex_disjoint_bfs_util(){

	// parallelise these functions
	clear_visited();
	clear_bfs_parent();
	clear_is_parent_change();

	// clear_frontier(my_frontier, my_next_frontier );
	initialise_partner_vertex();


	// int num_aug_paths = 1000;

	int tid = blockIdx.x*1024 + threadIdx.x;


	int vertex = tid+1;
	if(vertex > num_vertices1)
		return;

	if(!d_visited[vertex] && !d_is_matched_vertex[vertex]){
		d_frontier[vertex] = 1;
		vertex_disjoint_bfs(0, vertex, tid);
		__syncthreads();
	}


	if(num_aug_paths > 0){
		update_matchings();
	}

	// return num_aug_paths;
}

int check_matching(){
	int total_matched = 0;
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		int vertex = i;
		int num_matched = 0;


		for(int j=h_list_ptr[i];j<h_list_ptr[i+1];j++){
			int neighbor = h_flat_adj_list[j];
			// cout << vertex << " " << neighbor << endl;
			if(get_is_matched_edge_h(vertex, neighbor)){
				num_matched++;
			}
		}


		if(num_matched==1){
			// cout << "Hi" << endl;
			total_matched++;
		}
		if(num_matched>1){
			cout << vertex << endl;
			cout << "Error! Not a matching!";
			exit(0);
		}
	}
	return total_matched/2;
}


int main(){
	h_is_matched_edge = (bool *)calloc( (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1), sizeof(bool));

	h_flat_adj_list = (int *)malloc(2*num_edges*sizeof(int));
	h_degree = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_list_ptr = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_list_ptr_copy = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_is_matched_vertex = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_partner_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_visited = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_bfs_parent = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_is_parent_change = (bool *)malloc((num_vertices1+num_vertices2+1)*sizeof(bool));
	h_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_next_frontier = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));

	// Add a check for null memory

	memset(h_degree, 0, num_vertices1 + num_vertices2 +1);
	// memset(h_is_matched_edge, 0, (num_vertices1 + num_vertices2 +1)*(num_vertices1+num_vertices2+1));
	memset(h_is_matched_vertex, 0, num_vertices1 + num_vertices2 +1);
	memset(h_visited, 0, num_vertices1 + num_vertices2 +1);
	memset(h_is_parent_change, 0, num_vertices1 + num_vertices2 +1);
	memset(h_frontier, 0, num_vertices1 + num_vertices2 +1);
	memset(h_next_frontier, 0, num_vertices1 + num_vertices2 +1);



	// to and from of edges
	// int h_edges_u[num_edges], h_edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	int *h_edges_u, *h_edges_v;
	h_edges_u = (int *)malloc((num_edges)*sizeof(int));
	h_edges_v = (int *)malloc((num_edges)*sizeof(int));


	ifstream fin;
    // fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    fin.open("random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt", ios::in);
    cout << "random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt" <<endl;
    int u, v;

    // cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
    		// cout << i << endl;
            fin >> u >> v;
            h_edges_u[i] = u;
            h_edges_v[i] = v;
            h_degree[u]++;
            h_degree[v]++;
    }

    cout << "Done reading edges" << endl;

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    h_list_ptr[1] = 0;
    h_list_ptr_copy[1] = h_list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	h_list_ptr[i] = h_list_ptr[i-1] + h_degree[i-1];
    	h_list_ptr_copy[i] = h_list_ptr[i];
    }
    h_list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    h_list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position

    
    for(int i=0;i<num_edges;i++){
    	h_flat_adj_list[h_list_ptr_copy[h_edges_u[i]]] = h_edges_v[i];
    	h_flat_adj_list[h_list_ptr_copy[h_edges_v[i]]] = h_edges_u[i];
    	h_list_ptr_copy[h_edges_u[i]]++;
    	h_list_ptr_copy[h_edges_v[i]]++;
    }
    
	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_edge), h_is_matched_edge, (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), h_flat_adj_list, 2*num_edges*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_degree), h_degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), h_list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_is_matched_vertex), h_is_matched_vertex, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_visited), h_visited, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_frontier), h_frontier, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	
	// cudaMemcpy(d_is_matched_edge, h_is_matched_edge, (num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_flat_adj_list, h_flat_adj_list, 2*num_edges*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_degree, h_degree, (num_vertices1+num_vertices2+1)*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_list_ptr, h_list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_is_matched_vertex, h_is_matched_vertex, (num_vertices1+num_vertices2+1)*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_visited, h_visited, (num_vertices1+num_vertices2+1)*sizeof(int),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_frontier, h_frontier, (num_vertices1+num_vertices2+2)*sizeof(int),cudaMemcpyHostToDevice);


  	hipDeviceSynchronize();
  	vertex_disjoint_bfs_util<<<1, num_threads>>>();
// 1219611
  	hipDeviceSynchronize();
  	hipError_t bla = hipMemcpyFromSymbol(h_is_matched_edge, HIP_SYMBOL(d_is_matched_edge), sizeof(d_is_matched_edge),0,hipMemcpyDeviceToHost);
  	hipMemcpyFromSymbol(h_partner_vertex, HIP_SYMBOL(d_partner_vertex), sizeof(d_partner_vertex),0,hipMemcpyDeviceToHost);
  	hipDeviceSynchronize();
  

  	// for(int i=0;i<=neighborum_vertices1+num_vertices2;i++){
  	// 	cout << h_partner_vertex[i] << endl;
  	// }
  	// for(int i=0;i<)

  	int num_matches = check_matching();
  	printf("Number of matchings: %d", num_matches);

	
  	hipDeviceSynchronize();
  


}