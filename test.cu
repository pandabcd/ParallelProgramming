#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>

#define BLOCK_SIZE 2
#define GRID_SIZE 2

__global__ void test(int A[BLOCK_SIZE][BLOCK_SIZE], int B[BLOCK_SIZE][BLOCK_SIZE],int C[BLOCK_SIZE][BLOCK_SIZE])
{

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < BLOCK_SIZE && j < BLOCK_SIZE)
        C[i][j] = A[i][j] + B[i][j];
    printf("%d \n");

}

int main()
{

    int d_A[BLOCK_SIZE][BLOCK_SIZE];
    int d_B[BLOCK_SIZE][BLOCK_SIZE];
    int d_C[BLOCK_SIZE][BLOCK_SIZE];

    int C[BLOCK_SIZE][BLOCK_SIZE];

    for(int i=0;i<BLOCK_SIZE;i++)
      for(int j=0;j<BLOCK_SIZE;j++)
      {
        d_A[i][j]=i+j;
        d_B[i][j]=i+j;
      }


    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    // dim3 dimGrid(GRID_SIZE, GRID_SIZE); 

    // hipMemcpy(C,d_A,BLOCK_SIZE*BLOCK_SIZE , hipMemcpyDeviceToHost);
    // hipMemcpy(C,d_B,BLOCK_SIZE*BLOCK_SIZE , hipMemcpyDeviceToHost);
    // hipMemcpy(C,d_C,BLOCK_SIZE*BLOCK_SIZE , hipMemcpyDeviceToHost);
    
    test<<<dimGrid, dimBlock>>>(d_A,d_B,d_C); 



    // for(int i=0;i<BLOCK_SIZE;i++)
    //     {
    //       for(int j=0;j<BLOCK_SIZE;j++)
    //     {
    //       printf("%d\n",C[i][j]);

    //     }
    // }
}