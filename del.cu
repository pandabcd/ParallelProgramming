#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>
#include<bits/stdc++.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 

    hipDeviceSynchronize();
    return 0;
}