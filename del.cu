#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

// thrust::host_vector<thrust::device_vector<int>> b;

__global__
void func_check(int** b){
    printf("size of vector is %d", b[0][0]);
}


int main(){
    thrust::host_vector<int> a;
    a.push_back(100);
    thrust::device_vector<thrust::device_vector<int>> b;
    b.push_back(a)

    cout << a.size();
    // thrust::device_ptr<int> dp = b.data();
    int **dp = thrust::raw_pointer_cast(b.data());
    func_check<<<1,1>>>(dp);
}





// __global__
// void func_check(){
// 	printf("size of vector is %d", b.size());
// }


// int main(){
// 	thrust::host_vector<thrust::host_vector<int>> a;


// 	thrust::host_vector<int> temp;
// 	temp.push_back(1);
// 	temp.push_back(2);
// 	a.push_back(temp);
// 	temp.clear();
// 	temp.push_back(3);
// 	a.push_back(temp);

// 	thrust::host_vector<thrust::device_vector<int>> b = a;

// 	for(int i=0;i<a.size();i++){
// 		for(int j=0;j<a[i].size();j++){
// 			cout<<b[i][j]<<endl;
// 		}
// 		cout<<endl;
// 	}

// 	func_check<<<1,1>>>();

// }