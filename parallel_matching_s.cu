#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<fstream>
#include<vector>

using namespace std;

// #define num_threads 50
// #define num_edges 700000
// #define num_vertices1 10000
// #define num_vertices2 10000

// #define num_edges 1000000
// #define num_vertices1 1000
// #define num_vertices2 1000


#define num_edges 2998468
// #define num_vertices1 100000
// #define num_vertices2 100000

const long long num_vertices1 = 100000;
const long long num_vertices2 = 100000;

#define long long int lli
// vector<int> adj_list[num_vertices1 + num_vertices2 + 1];			// Do we need this? YES
	// vector<bool> is_matched_edge[num_vertices1 + num_vertices2 + 1];    // Adjacency matrix with boolean indicators
	// bool is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	// Is the vertex matched
	// int partner_vertex[num_vertices1 + num_vertices2 + 1];				// Get the vertex with which this vertex is matched. Initialised as -1  

	// int visited[num_vertices1+num_vertices2+1] = {0} ;			// Visited array for each vertex
	// int bfs_parent[num_vertices1+num_vertices2+1] ;				// Parent of the vertex. Required to find the augmenting path
	// int is_parent_change[num_vertices1+num_vertices2+1] = {0};	// Denotes if the parent changed in the last round
	// int num_aug_paths = 0;										// Counts number of augmenting paths found
int h_fc = num_vertices1;

int h_flat_adj_list[2*num_edges];
int h_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
int h_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
int h_list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep

// bool h_is_matched_edge[(num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)] = {0} ;     // Adjacency matrix (1-indexed)

bool *h_is_matched_edge;
bool h_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
int h_partner_vertex[num_vertices1 + num_vertices2 + 1];
int h_visited[num_vertices1 + num_vertices2 + 1] = {0};
int h_bfs_parent[num_vertices1 +  num_vertices2 + 1];
bool h_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

int fc = num_vertices1;
int num_aug_paths = 0;


// Only required for results
// int matched_vertices[num_vertices1+num_vertices2+1]={0};
// int matched_edges[2*num_edges]={0};

// vector<int> frontier;
// int aug_path_end = -1;
int frontier[num_vertices1 + num_vertices2] = {0};
int next_frontier[num_vertices1+num_vertices2] = {0};


int get_is_matched_edge(int i, int j){
	// cout << i << " " << j << endl;
	return h_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ];
}

void set_is_matched_edge(int i, int j, int value){
	h_is_matched_edge[i*(num_vertices1 + num_vertices2+1) + j ] = value;
}
// Checks if the matching is correct and also returns the total number of vertices matched
int check_matching(){
	int total_matched = 0;
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		int vertex = i;
		int num_matched = 0;
		int start_edge = h_list_ptr[vertex];
		int end_edge = h_list_ptr[vertex+1];

		for(int j=start_edge;j<end_edge;j++){

			int neighbor = h_flat_adj_list[j];
			// cout << "vertex-neighbor " << vertex << " " <<neighbor <<endl;
			if(get_is_matched_edge(vertex, neighbor)){
				// cout << "Matched" << endl;
				// cout << vertex << " " << neighbor <<endl;
				num_matched++;
			}
		}
		if(num_matched==1){
			total_matched++;
		}
		if(num_matched>1){
			// cout << vertex << endl;
			// cout << "Error! Not a matching!";
			// exit(0);
		}
	}
	cout << "Matching is correct! " << endl;
	return total_matched/2;
}

void clear_visited(){
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		h_visited[i] = 0;
	}
}

void clear_bfs_parent(){
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		h_bfs_parent[i] = i;
	}
}

void initialise_partner_vertex(){
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		h_partner_vertex[i] = -1;
	}
}

void clear_is_parent_change(){
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		h_is_parent_change[i] = 0;
	}
}





void print_matchings(){
	cout << "Matchings: " << endl;
    for(int i=1;i<=num_vertices1+num_vertices2; i++){
    	cout<< i << " " << h_partner_vertex[i] << endl;
    }
}

void match_edges(int u, int v){
	// h_is_matched_edge[u][v] = 1;
	// h_is_matched_edge[v][u] = 1;
	// cout << "Matching " << u << " " << v << endl;	

	set_is_matched_edge(u,v,1);
	set_is_matched_edge(v,u,1);
	h_is_matched_vertex[u] = 1;
	h_is_matched_vertex[v] = 1;
	h_partner_vertex[u] = v;
	h_partner_vertex[v] = u;
}

// Unmatching edges also unmatches the vertices since the graph is a matching
void unmatch_edges(int u, int v){
	// h_is_matched_edge[u][v] = 0;
	// h_is_matched_edge[v][u] = 0;
	

	// cout << "UnMatching " << u << " " << v << endl;
	set_is_matched_edge(u,v,0);
	set_is_matched_edge(v,u,0);
	h_is_matched_vertex[u] = 0;
	h_is_matched_vertex[v] = 0;
	h_partner_vertex[u] = -1;
	h_partner_vertex[v] = -1;
}


void update_matchings(){
	for(int i=1;i<=num_vertices1+num_vertices2;i++){
		int vertex = i;
		if(h_is_parent_change[vertex] == true){
			
			// cout << "Found aug. path till " << vertex << endl;
			// There should always be odd number of vertices in aug. path
			int path_length = 1;
			int parent = h_bfs_parent[vertex];
			while(parent!=vertex){
				// cout << vertex << " " <<parent << endl;
				if(path_length%2==1){
					match_edges(vertex, parent);
					// cout << "Matching " << vertex <<  " and " << parent << endl; 
				}
				else{
					unmatch_edges(vertex, parent);
					// cout << "Unmatching " << vertex <<  " and " << parent << endl;
				}
				vertex =  h_bfs_parent[vertex];
				parent = h_bfs_parent[vertex];
				path_length++;
				// cout << vertex << " " << parent << endl;
			}
			
			// cout << ". The path length is: " << path_length << endl;
			// break;
		}

		// return here to stop after updating only one path : Important for experiments
	}
}


int get_frontier_element(int ele){
	for(int i=ele+1;i<=num_vertices1+num_vertices2+1;i++){
		if(frontier[i]){
			return i;
		}
	}
	return -1;
}

void copy_frontier(){
	for(int i=0;i<=num_vertices1+num_vertices2;i++){
		frontier[i] = next_frontier[i];
		next_frontier[i] = 0;
	}
}

void bfs(bool binary_level){

	// vector<int> next_frontier;
	
	int frontier_element = get_frontier_element(0);
	// int frontier_element = 9265;

	// cout << "Frontier: " << frontier_element << endl;
	// if(not frontier.empty()){
	if(frontier_element!=-1){
		// for(int i=0;i<frontier.size();i++){
		// Iterate all frontier elements
		while(frontier_element!=-1){

			// int vertex = frontier[i];
			int vertex = frontier_element;
			h_visited[vertex] = true;
			
			// cout << "Frontier: " << frontier_element << endl;
			// cout << "Continuining for vertex: " << vertex << endl;
			bool found_path = false;
			int start_edge = h_list_ptr[vertex];
			int end_edge = h_list_ptr[vertex + 1]; 
			
			// cout << "Start-End edge " <<  start_edge << " " << end_edge  << endl;
			for(int j=start_edge;j<end_edge;j++){
				if(found_path)
					break;


				int neighbor = h_flat_adj_list[j];

				// cout << "Vertex- neighbor " << vertex << " " << neighbor <<endl; 

				if(!h_visited[neighbor]){
					// We want to alternate between unmatched and matched edges, otherwise we ignore
					h_visited[neighbor] = true;
					// cout << "Processing: " << vertex << " " << neighbor << endl;
					// exit(0);
					h_bfs_parent[neighbor] = vertex;

					if( binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && h_is_matched_vertex[neighbor]==1 ){
						// next_frontier.push_back(neighbor);
						next_frontier[neighbor] = 1;
					}

					// is_matched_vertex is implicitly true since the edge is matched
					// In level 1, we are only interested in matched edges
					else if( binary_level==1 && get_is_matched_edge(vertex, neighbor)==1 ){
						// next_frontier.push_back(neighbor);
						next_frontier[neighbor] = 1;
						// If I have found a path to the next level; I have to break
						// found_path = 1;
						return;
					}

					// Changing parent change only for this node
					else if(binary_level==0 && get_is_matched_edge(vertex, neighbor)==0 && h_is_matched_vertex[neighbor]==0){
						// cout << "Found a aug. path with " << neighbor << " with parent: " << vertex << endl;
						h_is_parent_change[neighbor] = 1;
						num_aug_paths++ ;
						// remove this return so that multiple paths can be found 
						return;
					}
				}
			}

			frontier_element = get_frontier_element(vertex);
		}
		// frontier.clear();
		// frontier.assign(next_frontier.begin(), next_frontier.end());
		copy_frontier();
		bfs(binary_level = !binary_level);
	}
	
}

void clear_frontier(){
	for(int i=0;i<num_vertices1+num_vertices2+1;i++){
		frontier[i] = 0;
	}
}

int bfs_util(){
	clear_visited();
	clear_bfs_parent();
	clear_is_parent_change();
	// frontier.clear();
	clear_frontier();

	//Can add fairness here

	num_aug_paths = 0;

	// Special style bfs
	for(int i=1;i<=num_vertices1;i++){
		if(!h_visited[i] && !h_is_matched_vertex[i]){
			// frontier.clear();
			clear_frontier();
			// frontier.push_back(i);
			frontier[i] = 1;
			bfs(0);
			// cout << "Loop";
		}
		// break;	
	}

	// cout << "Printing parents: " << endl;
	// for(int i=1;i<=num_vertices2+num_vertices1;i++){
	// 	cout << i << " " << h_bfs_parent[i] <<endl;
	// }

	if(num_aug_paths > 0){
		update_matchings();
	}

	return num_aug_paths;

}


int main(){
	cout << "hi " << endl;
	h_is_matched_edge = (bool *)malloc((num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)*sizeof(int));
	h_flat_adj_list = (int *)malloc(2*num_edges*sizeof(int));
	h_list_ptr = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_list_ptr_copy = (int *)malloc((num_vertices1+num_vertices2+2)*sizeof(int));
	h_is_matched_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_partner_vertex = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_visited = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_bfs_parent = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));
	h_is_parent_change = (int *)malloc((num_vertices1+num_vertices2+1)*sizeof(int));


// int h_flat_adj_list[2*num_edges];
// int h_degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
// int h_list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
// int h_list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep

// // bool h_is_matched_edge[(num_vertices1+ num_vertices2 + 1)*(num_vertices1 + num_vertices2+1)] = {0} ;     // Adjacency matrix (1-indexed)

// bool h_is_matched_vertex[num_vertices1 + num_vertices2 + 1] = {0};	//is the vertex matched
// int h_partner_vertex[num_vertices1 + num_vertices2 + 1];
// int h_visited[num_vertices1 + num_vertices2 + 1] = {0};
// int h_bfs_parent[num_vertices1 +  num_vertices2 + 1];
// bool h_is_parent_change[num_vertices1 + num_vertices2 + 1] = {0};

	

	// to and from of edges
	int h_edges_u[num_edges], h_edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	

	
	ifstream fin;
    // fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    fin.open("random_" + to_string(num_vertices1) + "_" + to_string(num_vertices2) + ".txt", ios::in);
    int u, v;

    // cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
            fin >> u >> v;
            // cout << u << " " << v <<endl;
            h_edges_u[i] = u;
            h_edges_v[i] = v;
            h_degree[u]++;
            h_degree[v]++;
    }

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    h_list_ptr[1] = 0;
    h_list_ptr_copy[1] = h_list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	h_list_ptr[i] = h_list_ptr[i-1] + h_degree[i-1];
    	h_list_ptr_copy[i] = h_list_ptr[i];
    }
    h_list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    h_list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position



    for(int i=0;i<num_edges;i++){
    	h_flat_adj_list[h_list_ptr_copy[h_edges_u[i]]] = h_edges_v[i];
    	h_flat_adj_list[h_list_ptr_copy[h_edges_v[i]]] = h_edges_u[i];
    	h_list_ptr_copy[h_edges_u[i]]++;
    	h_list_ptr_copy[h_edges_v[i]]++;
    }


    // for(int i=1;i<=num_vertices1+num_vertices2;i++){
    // 	for(int j=1;j<=num_vertices1+num_vertices2+1;j++){
    // 		h_is_matched_edge[j*num_vertices2 + i] = 0;
    // 	}
    // }


    initialise_partner_vertex();
  	

  	// for(int i=1;i<=num_vertices1+num_vertices2;i++){
  	// 	cout << h_degree[i] << " ";
  	// }

  	// for(int i=0;i<2*num_edges;i++){
  	// 	cout << h_flat_adj_list[i] << " ";
  	// }
  	// cout << endl;
   //  for(int i=0;i<=num_vertices1+num_vertices2;i++){
  	// 	cout << h_list_ptr[i] << " ";
  	// // }

    // cout << " ------------------------" <<endl;
    // for(int i=1;i<=num_vertices1;i++){
    // 	for(int j=1;j<=num_vertices2;j++){
    // 		get_is_matched_edge(i,j);
    // 	}
    // }


    // cout << get_frontier_element(9265);
  	int x = check_matching();
    bfs_util();
    print_matchings();


    x = check_matching();
    cout << "Number of matchings: " << x << endl;








    // int x = check_matching();
    // cout << "Total matches before running code: " << x << endl;
    
    
    // int aug_paths = bfs_util();
    // cout << "Main : Number of augmenting paths " << aug_paths << endl;
    // // print_matchings();

    // while(aug_paths>0)
    // {	
    // 	aug_paths = bfs_util();
    // 	cout << "Main : Number of augmenting paths " << aug_paths << endl;
    // 	// print_matchings();
    // 	break;
    // }

    // x = check_matching();
    // cout << "Total matches " << x/2 << endl;


}