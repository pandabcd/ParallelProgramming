#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

#define num_threads 50
#define num_edges 25
#define num_vertices1 5
#define num_vertices2 5

__device__ unsigned int d_degree[num_vertices1+num_vertices2+1];    //Is this required?
__device__ unsigned int d_flat_adj_list[2*num_edges];
__device__ unsigned int d_list_ptr[num_vertices1+num_vertices2+2];

__device__ unsigned int d_matched_vertices[num_vertices1+num_vertices2+1]={0};
__device__ unsigned int d_matched_edges[2*num_edges]={0};
__device__ unsigned int d_visited[num_vertices1+num_vertices2+1]={0};
		


// Every vertex gets a node
__global__ 
void get_approx_matching(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex = tid + 1;	// The world is 1-indexed
	if(vertex<=num_vertices1){

		printf("[%d]Looking from %d to %d \n" ,tid, d_list_ptr[vertex], d_list_ptr[vertex+1]);
		for(int i=d_list_ptr[vertex];i<d_list_ptr[vertex+1];i++){


			// Problem in here.... You can do it :)
			printf("[%d]working %d \n",tid, d_list_ptr[vertex]);
			int visited = atomicExch(&d_visited[d_list_ptr[vertex]], 1);    // Index of connected vertex
			printf("inside %d \n", visited);
			if(!visited)
			{
				printf("Pairing %d with %d \n", vertex, d_flat_adj_list[d_list_ptr[vertex]]);
				// d_matched[i] = 1;
				return;
			}
		}

	}
}


__global__
void vertex_disjoint_bfs(){

}


int main(){
	int fc = num_vertices1;
	
	int degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
	int flat_adj_list[2*num_edges];
	int list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
	int list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep
	// Only required for results
	int matched_vertices[num_vertices1+num_vertices2+1]={0};
	int matched_edges[2*num_edges]={0};

	// to and from of edges
	int edges_u[num_edges], edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	

	
	ifstream fin;
    fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    int u, v;

    cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
            fin >> u >> v;
            cout << u << " " << v <<endl;
            edges_u[i] = u;
            edges_v[i] = v;
            degree[u]++;
            degree[v]++;
    }

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    list_ptr[1] = 0;
    list_ptr_copy[1] = list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	list_ptr[i] = list_ptr[i-1] + degree[i-1];
    	list_ptr_copy[i] = list_ptr[i];
    }
    list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;



    for(int i=0;i<num_edges;i++){
    	flat_adj_list[list_ptr_copy[edges_u[i]]] = edges_v[i];
    	flat_adj_list[list_ptr_copy[edges_v[i]]] = edges_u[i];
    	list_ptr_copy[edges_u[i]]++;
    	list_ptr_copy[edges_v[i]]++;
    }

    cout << "Printing flat adjacency list for 4: " << endl;
    // for(int i=0;i<2*num_edges;i++){
    // 	cout << flat_adj_list[i] << endl;
    // }

    for(int i=list_ptr[4];i<list_ptr[5];i++){
    	cout << flat_adj_list[i] << endl;
    }

   
    hipMemcpyToSymbol(HIP_SYMBOL(d_degree), degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), flat_adj_list, (2*num_edges)*sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	
    // cout<< list_ptr[0];
    cout<<endl<<endl;
	get_approx_matching<<<1, num_threads>>>();

	// hipMemcpyFromSymbol(matched, HIP_SYMBOL(d_matched), num_edges*sizeof(int), 0, hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	// cout << "Printing matched edges"<<endl;
	// for(int i=0;i<num_edges;i++){
	// 	if(matched[i]){
	// 		cout << edges_u[i] << " " << edges_v[i] << endl;
	// 	}
	// }

	
	return 0;
}