#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>


// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

using namespace std;

#define num_threads 50
#define num_edges 3
#define num_vertices1 2
#define num_vertices2 2

__device__ const int frontier_size = 5;    // Maximum of size of num_vertices1, num_vertices26h


// Some of these can go to constant memory, check that
// But constant memory is 65KB while global memory is 4040MB, so there is that limitation
__device__ unsigned int d_degree[num_vertices1+num_vertices2+1];    //degree of vertices  //Is this required?
__device__ unsigned int d_flat_adj_list[2*num_edges];				//adjacency list flattened
__device__ unsigned int d_list_ptr[num_vertices1+num_vertices2+2];	//start indices of every vertex in adj_list

__device__ unsigned int d_matched_vertices[num_vertices1+num_vertices2+1]={0};	// whether the vertex is matched
__device__ unsigned int d_visited[num_vertices1+num_vertices2+1]={0};			// whether the vertex has been visited
__device__ unsigned int d_matched_with[num_vertices1+num_vertices2+1] = {0};
// __device__ unsigned int d_matched_edges[2*num_edges]={0};						//whether the edges is matched


// Every vertex gets a node
__global__ 
void get_approx_matching(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;	// The world is 1-indexed
	if(vertex1<=num_vertices1){

		for(int i=d_list_ptr[vertex1];i<d_list_ptr[vertex1+1];i++){
			int vertex2 = d_flat_adj_list[i];									// Index of connected vertex
			int visited = atomicExch(&d_visited[vertex2], 1);    
			if(!visited)
			{
				printf("Pairing %d with %d  which is index %d  \n", vertex1, vertex2, i);
				d_matched_vertices[vertex1] = 1;   // Marking the vertex as matched
				d_matched_vertices[vertex2] = 1;
				d_matched_with[vertex1] = vertex2;
				d_matched_with[vertex2] = vertex1;
				return;
			}
			
		}

	}
}


__device__
void clear_visited_list(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	int vertex1 = tid + 1;

	if(vertex1<=num_vertices1){
		d_visited[vertex1] = 0;
	}
}

// __global__
// void vertex_disjoint_bfs(){
// 	clear_visited_list();

// 	int tid = blockIdx.x*1024 + threadIdx.x;
// 	int vertex1 = tid + 1;

// 	if(vertex1<=num_vertices1){
// 		//If already matched
// 		if(d_matched_vertices[vertex1]==1){
// 			return;
// 		}

// 		// If already visited by some other thread
// 		int visited1 = atomicExch(&d_visited[vertex1], 1);
// 		if(visited1){
// 			return;
// 		}

// 		// If not already matched and no thread has visited this
// 		int frontiers[frontier_size];
// 	}
// }

//Vertices are 1-indexed(0th vertex will be source in future expansions) while adjacency list is 0 indexed
int main(){
	int fc = num_vertices1;
	
	int degree[num_vertices1+num_vertices2+1]={0};      //store degree of each vertex
	int flat_adj_list[2*num_edges];
	int list_ptr[num_vertices1+num_vertices2+2];        //1-indexed and extra element at the end for easy size access  // Pointer to the start of adjacency list
	int list_ptr_copy[num_vertices1+num_vertices2+2];    // Temporrary stuff, gotta sleep
	// Only required for results
	int matched_vertices[num_vertices1+num_vertices2+1]={0};
	int matched_edges[2*num_edges]={0};

	// to and from of edges
	int edges_u[num_edges], edges_v[num_edges];			// Make this dynamic memory and free it once we have our 2 pass initialisation phase
	

	
	ifstream fin;
    fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    int u, v;

    cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
            fin >> u >> v;
            cout << u << " " << v <<endl;
            edges_u[i] = u;
            edges_v[i] = v;
            degree[u]++;
            degree[v]++;
    }

    // Get pointer to adjacency list using prefix sum (no opti here since other parts are more complex anyway)
    // Index 0 will never be used.... the last elem
    list_ptr[1] = 0;
    list_ptr_copy[1] = list_ptr[1];
    for(int i=2;i<=num_vertices1+num_vertices2;i++){
    	list_ptr[i] = list_ptr[i-1] + degree[i-1];
    	list_ptr_copy[i] = list_ptr[i];
    }
    list_ptr[num_vertices1+num_vertices2+1] = 2*num_edges;       //For easy coding
    list_ptr_copy[num_vertices1+num_vertices2+1] = 2*num_edges;  // list_ptr has the start of the adj list ; list_ptr_copy has the current position



    for(int i=0;i<num_edges;i++){
    	flat_adj_list[list_ptr_copy[edges_u[i]]] = edges_v[i];
    	flat_adj_list[list_ptr_copy[edges_v[i]]] = edges_u[i];
    	list_ptr_copy[edges_u[i]]++;
    	list_ptr_copy[edges_v[i]]++;
    }

    // cout << "Printing flat adjacency list: " << endl;
    // for(int i=0;i<2*num_edges;i++){
    // 	cout << flat_adj_list[i] << endl;
    // }

    // for(int i=list_ptr[4];i<list_ptr[5];i++){
    // 	cout << flat_adj_list[i] << endl;
    // }

   
    hipMemcpyToSymbol(HIP_SYMBOL(d_degree), degree, (num_vertices1+num_vertices2+1)*sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_flat_adj_list), flat_adj_list, (2*num_edges)*sizeof(int),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_list_ptr), list_ptr, (num_vertices1+num_vertices2+2)*sizeof(int),0,hipMemcpyHostToDevice);
	
    // cout<< list_ptr[0];
    cout<<endl<<endl;
	get_approx_matching<<<1, num_threads>>>();
	// vertex_disjoint_bfs<<<1, num_threads>>>();   // Call this inside the first kernel call only

	// hipMemcpyFromSymbol(matched, HIP_SYMBOL(d_matched), num_edges*sizeof(int), 0, hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	// cout << "Printing matched edges"<<endl;
	// for(int i=0;i<num_edges;i++){
	// 	if(matched[i]){
	// 		cout << edges_u[i] << " " << edges_v[i] << endl;
	// 	}
	// }

	
	return 0;
}