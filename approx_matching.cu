#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>


using namespace std;

#define num_threads 50
#define num_edges 25
#define num_vertices1 5
#define num_vertices2 5


// Stores edges
__device__ int d_edges_u[num_edges];
__device__ int d_edges_v[num_edges];

// __device__ unsigned int d_visited_1[num_vertices1]={0};

__device__ unsigned int d_visited_2[num_vertices2]={0};  //visited or not
__device__ unsigned int d_matched[num_edges]={0};		// matched or not

__device__ unsigned int d_first_edge[num_vertices1+1];  // making vertex disjoint


__global__ 
void get_approx_matching(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	if(tid<num_edges){
		// printf("[%d] %d \n", tid, d_first_edge[tid]);
		for(int i=d_first_edge[tid]; i<d_first_edge[tid+1];i++)
		{
			int visited2 = atomicExch(&d_visited_2[d_edges_v[i]], 1);
			if(!visited2)
			{
				printf("Pairing %d with %d (edge number %d)\n", d_edges_u[i], d_edges_v[i], i);
				d_matched[i] = 1;
				return;
			}
		}
	}
	
}


int main(){
	int fc = num_vertices1;
	int set1[num_vertices1], set2[num_vertices2];
	int first_edge[num_vertices1+1]; // one artifial index at last for easier coding
	int edges_u[num_edges], edges_v[num_edges];
	int matched[num_edges]={0};

	ifstream fin;
    fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    int u, v;

    for(int i=0;i<num_vertices1;i++){
    	fin >> first_edge[i];
    }
    first_edge[num_vertices1] = num_edges;

    cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
            fin >> u >> v;
            // Check if not a repeat, then add
            // set1.push_back(u);
            // set2.push_back(v);
            edges_u[i] = u;
            edges_v[i] = v;
            cout << u << " " << v <<endl;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_first_edge), first_edge, (num_vertices1+1)*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_edges_u), edges_u, num_edges*sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_edges_v), edges_v, num_edges*sizeof(int), 0, hipMemcpyHostToDevice);


	get_approx_matching<<<1, num_threads>>>();

	hipMemcpyFromSymbol(matched, HIP_SYMBOL(d_matched), num_edges*sizeof(int), 0, hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	cout << "Printing matched edges"<<endl;
	for(int i=0;i<num_edges;i++){
		if(matched[i]){
			cout << edges_u[i] << " " << edges_v[i] << endl;
		}
	}

	
	return 0;
}