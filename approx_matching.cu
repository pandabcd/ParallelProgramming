#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_cooperative_groups.h>
#include<hip/hip_runtime_api.h>


using namespace std;

#define num_threads 50
#define num_edges 4
#define num_vertices1 2
#define num_vertices2 2


__device__ int d_edges_u[num_edges];
__device__ int d_edges_v[num_edges];

__device__ unsigned int d_visited_1[num_vertices1]={0};
__device__ unsigned int d_visited_2[num_vertices2]={0};
__device__ unsigned int d_matched[num_edges]={0};

__device__ unsigned int d_first_edge[num_vertices1];

__global__ 
void get_approx_matching(){
	int tid = blockIdx.x*1024 + threadIdx.x;
	if(tid<num_edges){

		int visited1 = atomicExch(&d_visited_1[d_edges_u[tid]], 1);   
		if(!visited1){
			printf("[%d]%d is unvisitedd \n", tid,d_edges_u[tid]);

			int visited2 = atomicExch(&d_visited_2[d_edges_v[tid]], 1);
			if(!visited2)
			{
				   
				printf("[%d]%d is unvisited::::: %d \n",tid, d_edges_v[tid], visited2);
				printf("Pairing %d with %d \n", d_edges_u[tid], d_edges_v[tid]);
			}
			else{
				printf("[%d]%d is visitedd \n",tid, d_edges_v[tid]);

			}
		}
	}	
}


int main(){
	int fc = num_vertices1;
	int set1[num_vertices1], set2[num_vertices2];
	int first_edge[num_vertices1];
	int edges_u[num_edges], edges_v[num_edges];

	ifstream fin;
    fin.open("FC_" + to_string(fc) + "_" + to_string(fc) + ".txt", ios::in);
    int u, v;

    for(int i=0;i<fc;i++){
    	fin >> first_edge[i];
    }

    cout << "Printing all the edges: \n";

    // Vertices with 0 edges are implicitly ignored while reading the file itself
    for(int i=0;i<num_edges;i++){
            fin >> u >> v;
            // Check if not a repeat, then add
            // set1.push_back(u);
            // set2.push_back(v);
            edges_u[i] = u;
            edges_v[i] = v;
            cout << u << " " << v <<endl;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_first_edge), first_edge, num_vertices1*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_edges_u), edges_u, num_edges*sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_edges_v), edges_v, num_edges*sizeof(int), 0, hipMemcpyHostToDevice);


	get_approx_matching<<<1, num_threads>>>();

	hipDeviceSynchronize();
	return 0;
}